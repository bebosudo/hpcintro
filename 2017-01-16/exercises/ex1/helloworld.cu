#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void hello();

int main(int argc, char const *argv[]) {

    hello<<<100, 100>>>();
    // checkCudaErrors(hipDeviceSynchronize());
    hipDeviceSynchronize();

    return 0;
}


__global__ void hello() {
    if (blockIdx.x*blockDim.x+threadIdx.x == 100) {
        int* a = (int*) 0x10000; *a = 0;
    }

    printf("Hello! I'm thread %d out of %d in block %d. My global thread is "
            "%d out of %d.\n",
            threadIdx.x, blockDim.x, blockIdx.x,
            blockIdx.x*blockDim.x+threadIdx.x,
            gridDim.x*blockDim.x);
}
