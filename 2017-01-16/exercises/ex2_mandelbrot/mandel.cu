#include "hip/hip_runtime.h"
#include <stdio.h>
// #include <hip/hip_runtime_api.h>
#include "mandel.h"
#include "writepng.h"
#include <math.h>

__global__ void mandel(int disp_width, int disp_height,
                       int *array, int max_iter) {

    double 	scale_real, scale_imag;
    double 	x, y, u, v, u2, v2;
    int 	i, j, iter;

    scale_real = 3.5 / (double)disp_width;
    scale_imag = 3.5 / (double)disp_height;

    i = blockIdx.y * blockDim.y + threadIdx.y;
    j = blockIdx.x * blockDim.x + threadIdx.x;

    // If we are trying to access fields out of the array size, return.
    // if (col < disp_width) {
    if (i < disp_width && j < disp_height) {
    // for(i = 0; i < disp_width; i++)
        // printf("col: %d\n", col);

        // i = col; j = row;
        // i = row * gridDim.x * blockDim.x + col;
        x = ((double)i * scale_real) - 2.25;

        // for(j = 0; j < disp_height; j++) {

        y = ((double)j * scale_imag) - 1.75;

        u = v = u2 = v2 = 0.0;
        iter = 0;

        while ( u2 + v2 < 4.0 &&  iter < max_iter ) {
            v = 2 * v * u + y;
            u = u2 - v2 + x;
            u2 = u*u;
            v2 = v*v;
            iter = iter + 1;
        }

        // if we exceed max_iter, reset to zero
        iter = iter == max_iter ? 0 : iter;

        array[i*disp_width + j] = iter;
        // printf("%d \n", array[i*disp_height + j]);
    }
}
