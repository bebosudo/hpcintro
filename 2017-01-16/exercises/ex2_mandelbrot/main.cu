#include "hip/hip_runtime.h"
#include <stdio.h>
// #include <hip/hip_runtime_api.h>
#include "mandel.h"
#include "writepng.h"
#include <math.h>


int main(int argc, char const *argv[]) {

    int   width, height;
    int	  max_iter;
    int   *image;

    width = height = 2601;
    max_iter = 400;

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);

    // image = (int *)malloc( width * height * sizeof(int));
    hipMalloc(image, width * height * sizeof(int));
    if ( image == NULL ) {
       fprintf(stderr, "memory allocation for the image failed!\n");
       return(1);
    }

    // mandel(width, height, image, max_iter);

    // 32 is the number of threads for each GPUs
    int num_blocks = ceil(width/32.0);
    mandel<<<num_blocks, 32>>>(width, height, image, max_iter);

    // checkCudaErrors(hipDeviceSynchronize());
    hipDeviceSynchronize();

    writepng("mandelbrot.png", image, width, height);

    hipFree(image);
    
    return(0);
}
