#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "mandel.h"
#include "writepng.h"
#include <math.h>
#include <omp.h>


int main(int argc, char const *argv[]) {

    // just to avoid running the code on the 1st GPU (GPU=0) as everyone else.
    hipSetDevice(6);
    int   width, height;
    int	  max_iter;
    int   *h_image;
    int   *d_image;

    width = height = 2601;
    max_iter = 400;

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);

    int size_array = width * height * sizeof(int);

    // allocate space for two arrays on both the device and the host, but there's nothing to copy.
    h_image = (int *)malloc( size_array );
    checkCudaErrors(hipMalloc(&d_image, size_array ));
    if ( h_image == NULL || d_image == NULL ) {
       fprintf(stderr, "memory allocation for the image failed!\n");
       return(1);
    }

    // mandel(width, height, image, max_iter);

    // dim3 dimBlock(32);   // 1D
    // The nvidia GPUs we are working on allow us to spawn at most 1024 threads
    // per block, so we can use blocks of 32 threads for each side. 32**2=1024.
    dim3 dimBlock(32, 32);

    // In this way we round up to the greatest integer without having to do casts.
    // (width-31)/dimBlock.x
    dim3 dimGrid(((width+dimBlock.x-1)/dimBlock.x), ((height+dimBlock.y-1)/dimBlock.y));

    // --------------- MANDEL EXECUTION  ----------------
    double time = omp_get_wtime();
    mandel<<<dimGrid, dimBlock>>>(width, height, d_image, max_iter);
    checkCudaErrors(hipDeviceSynchronize());
    double elapsed = omp_get_wtime()-time;
    printf("\nmandel exec time = %lf\n", elapsed);
    // --------------------------------------------------


    // ----------------- COPY DtoH ----------------------
    double time_mem = omp_get_wtime();
    checkCudaErrors(hipMemcpy(h_image, d_image, size_array, hipMemcpyDeviceToHost));
    double elapsed_mem = omp_get_wtime()-time;
    printf("\ncopy DtoH time = %lf\n", elapsed_mem);
    // --------------------------------------------------


    writepng("mandelbrot.png", h_image, width, height);

    hipFree(d_image);
    free(h_image);

    return(0);
}
