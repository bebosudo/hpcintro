
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void jacobi(double * uold, double * unew, double * f, int N, double lambda2){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if(index >= N*N){
		return;
	}
//	printf("index: %d, index div N: %d\n", index, index/N);


	int M = N+2;
	int i = index + M + 1 + 2 * (index / N);

	unew[i] = 0.25 * (uold[i-1] + uold[i+1] + uold[i-M] + uold[i+M] + lambda2*f[i]);

}
