
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void jacobi(double * uold, double * unew, double * f,int N, double lambda2);

int main(int argc, char * argv[]){
	if (argc != 3){
		printf("Wrong number of argument (N kmax)");
		return -1;
	}

	int N = atoi(argv[1]);
	int M = N+2;
	int kmax = atoi(argv[2]);
	double lambda2 = (double)4/(M*M);

	int size = M*M*sizeof(double);

	double *u1 =(double *)calloc(M*M,sizeof(double));
	double *u2 = (double *)calloc(M*M,sizeof(double));
	double *f = (double *)calloc(M*M,sizeof(double));
	if (u1 == NULL || u2 == NULL || f == NULL){
		printf("Memory allocation failed");
		return -1;
	}

	for(int i = 0; i < N+1; i++){
		u2[i*M] = 20;
		u2[N+1+M*i] = 20;
		u2[i] = 20;
		u1[i*M] = 20;
		u1[(N+1)+i*M] = 20;
		u1[i] = 20;
		for (int i = 2*M/3; i <= 5*M/6; i++){
			for (int j = M/2; j <= 2*M/3; j++){
				f[i*M+j] = 200;
			}
		}
	}

	double *d_u1, *d_u2, *d_f;

	hipMalloc( (void**)&d_u1, size);
	hipMalloc( (void**)&d_u2, size);
	hipMalloc( (void**)&d_f, size);

	hipMemcpy( d_f, f, size, hipMemcpyHostToDevice );
	hipMemcpy( d_u1, u1, size, hipMemcpyHostToDevice );
	hipMemcpy( d_u2, u2, size, hipMemcpyHostToDevice );

	for(int k = 0; k<kmax; k++){
		jacobi<<<1,1>>>(d_u1,d_u2,d_f,N,lambda2);
		hipDeviceSynchronize();
	
		jacobi<<<1,1>>>(d_u2,d_u1,d_f,N,lambda2);
		hipDeviceSynchronize();
	}

	hipMemcpy( u1, d_u1, size, hipMemcpyDeviceToHost );
	
	FILE *fp1 = fopen("results.txt","w");
	if (fp1 == NULL) {
		printf("Error opening file\n");
		return -1;
	}

	for (int i = 0; i < M; i++){
		for (int j = 0; j < M; j++){
			fprintf(fp1,"%.2lf ",u1[i*M+j]);
		}
		fprintf(fp1,"\n");
	}

	hipFree(d_u1);
	hipFree(d_u2);
	hipFree(d_f);
	return 0;
}
