
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cblas.h"
#include <stdlib.h>

// version from the 1st assignment, adapted to use single pointer array access.
extern void matmult_lib(int m, int n, int k, double *A, double *B, double *C){
  double alpha = 1.0, beta = 0.0;
  cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans,
              m,n,k, alpha, A, k, B, n, beta, C, n);
}
