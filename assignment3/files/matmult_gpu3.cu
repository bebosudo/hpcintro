// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____           ____n____
//    |        |           |        |          |       |
//  m |    A   |   X    k  |    B   |  =    m  |   C   |
//    |        |           |        |          |       |
//    ---------            ---------           ---------

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>
//Version 1: second element below neighbor
__global__ void m3_1(int m, int n, int k, double *A, double *B, double *C) {

  double sum1 = 0,sum2 = 0;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  i *= 2;
  if (i < m && j < n){
      for (int h = 0; h < k; h++) {
        sum1 += A[i*k + h] * B[h*n + j];
        sum2 += A[(i+1)*k + h] * B[h*n + j];
      }
  C[i*n + j] = sum1;
  C[(i+1)*n + j] = sum2;
  }
}

//Version 2: second element right neighbor
__global__ void m3_2(int m, int n, int k, double *A, double *B, double *C) {

  double sum1 = 0,sum2 = 0;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  j *= 2;
  if (i < m && j < n){
      for (int h = 0; h < k; h++) {
        sum1 += A[i*k + h] * B[h*n + j];
        sum2 += A[i*k + h] * B[h*n + j+1];
      }
  C[i*n + j] = sum1;
  if (j+1 < n)
   C[i*n + j + 1] = sum2;
  }
}

//Version 3: second element element+blockDim
__global__ void m3_3(int m, int n, int k, double *A, double *B, double *C) {

  double sum1 = 0,sum2 = 0;
  int i = blockIdx.x*blockDim.x*2+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  if (i < m && j < n){
      for (int h = 0; h < k; h++) {
        sum1 += A[i*k + h] * B[h*n + j];
        sum2 += A[(i+blockDim.x)*k + h] * B[h*n + j];
      }
  C[i*n + j] = sum1;
  C[(i+blockDim.x)*n + j] = sum2;
  }
}

extern "C" {
    void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipSetDevice(2);
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        // Initialize the output matrix with zeroes.
        hipMemset(d_C, 0, m*n * sizeof(double));
        dim3 BlockDim(16,16);
        dim3 NumBlocks((m/2-1)/16+1,((n-1)/16+1));

        double time = omp_get_wtime();
        m3_1<<<NumBlocks,BlockDim>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();
        double elapsed1 = omp_get_wtime() - time;

        printf("Kernel: %lf s\n",elapsed1);

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
