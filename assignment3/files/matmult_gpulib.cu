// #include <stdio.h>

// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____          ____n____
//    |        |           |        |         |       |
//  m |   A    |    X    k |   B    |  =   m  |   C   |
//    |        |           |        |         |       |
//    ---------            ---------          ---------

#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "stdio.h"

extern "C" {
  void matmult_gpulib(int m, int n, int k, double *A, double *B, double *C) {
    hipSetDevice(2);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0, beta = 0.0;

    double* d_A, * d_B, * d_C;
    hipMalloc((void**) &d_A, m*k * sizeof(double));
    hipMalloc((void**) &d_B, k*n * sizeof(double));
    hipMalloc((void**) &d_C, m*n * sizeof(double));
    hipMemcpy(d_A, A,  m*k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B,  k*n * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_C, 0,  m*n * sizeof(double));

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, &d_A[0], k, &d_B[0], n, &beta, &d_C[0], n);

    hipblasDestroy(handle);

    hipMemcpy(C, d_C,  m*n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

  }
}
