
#include <hip/hip_runtime.h>
// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____           ____n____
//    |        |           |        |          |       |
//  m |    A   |   X    k  |    B   |  =    m  |   C   |
//    |        |           |        |          |       |
//    ---------            ---------           ---------

__global__ void m4_1(int m, int n, int k, double *A, double *B, double *C) {
  double sum1 = 0, sum2 = 0, sum3 = 0, sum4 = 0;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  j *= 4;
  if (i < m && j < n){
      for (int h = 0; h < k; h++) {
        sum1 += A[i*k + h] * B[h*n + j];
        if (j+1 < n) sum2 += A[i*k + h] * B[h*n + j+1];
        if (j+2 < n) sum3 += A[i*k + h] * B[h*n + j+2];
        if (j+3 < n) sum4 += A[i*k + h] * B[h*n + j+3];
      }
  C[i*n + j] = sum1;
  if (j+1 < n) C[i*n + j+1] = sum2;
  if (j+2 < n) C[i*n + j+2] = sum3;
  if (j+2 < n) C[i*n + j+3] = sum4;
  }
}


__global__ void m4_2(int m, int n, int k, double *A, double *B, double *C) {
  double sum1 = 0, sum2 = 0, sum3 = 0, sum4 = 0;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  i *= 4;
  if (i < m && j < n){
      for (int h = 0; h < k; h++) {
        sum1 += A[i*k + h] * B[h*n + j];
        sum2 += A[(i+1)*k + h] * B[h*n + j];
        sum3 += A[(i+2)*k + h] * B[h*n + j];
        sum4 += A[(i+3)*k + h] * B[h*n + j];
      }
  C[i*n + j] = sum1;
  C[(i+1)*n + j] = sum2;
  if (i+3 < m) C[(i+2)*n + j] = sum3;
  if (i+3 < m) C[(i+3)*n + j] = sum4;
  }
}

extern "C" {
    void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        // Initialize the output matrix with zeroes.
        hipMemset(d_C, 0, m*n * sizeof(double));
        dim3 BlockDim(16,16);
        dim3 NumBlocks((m-1)/16+1,(((n-1)/4+1)/16+1));
        m4_1<<<NumBlocks,BlockDim>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
