#include "hip/hip_runtime.h"
// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____          ____n____
//    |        |           |        |         |       |
//  m |   A    |    X    k |   B    |  =   m  |   C   |
//    |        |           |        |         |       |
//    ---------            ---------          ---------


__global__ void m5(int m, int n, int k, double *A, double *B, double *C) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    // This variable 'two_blocks' (the name can be changed to whatever) comes
    // from the kernel invocation, and we have to "split" it manually into the
    // two variables we want to use.
    extern __shared__ double two_blocks[];
    double* A_s = &two_blocks[0];
    double* B_s = &two_blocks[blockDim.x*blockDim.y];

    if (i < m && j < n) {
        A_s[threadIdx.y*blockDim.x + threadIdx.x] = A[i*n + j];
        B_s[threadIdx.x*blockDim.y + threadIdx.y] = B[i*n + j];

        __syncthreads();

        int ii = threadIdx.y;
        int jj = threadIdx.x;
        double sum = 0.0;

        for (int h = 0; h < blockDim.y; h++) {
            sum += A_s[ii*blockDim.y + h] * B_s[h*n + jj];
        }
        C[i*n + j] += sum;
    }
}


extern "C" {
    void matmult_gpu5(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        // Initialize the output matrix with zeroes.
        hipMemset(d_C, 0, m*n * sizeof(double));

        dim3 blockDim(16,16);
        dim3 gridDim( (m-1)/blockDim.x+1, (n-1)/blockDim.y+1 );


        // https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
        // dynamically "pass" the shared memory to the kernel function.
        // Otherwise we should place some constants in the kernel function.
        m5<<<blockDim, gridDim, (blockDim.x*blockDim.y * 2 * sizeof(double))>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
Contact GitHub API Training Shop Blog About
© 2017 GitHub, Inc. Terms Privacy Security Status Help
