#include "hip/hip_runtime.h"
// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____           ____n____
//    |        |           |        |          |       |
//  m |    A   |   X    k  |    B   |  =    m  |   C   |
//    |        |           |        |          |       |
//    ---------            ---------           ---------
#include <hip/hip_runtime_api.h>
__global__ void m6(int m, int n, int k, double *A, double *B, double *C) {
  double sum;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
  __shared__ double A_s[16*16];
  __shared__ double B_s[16*16];
  int ii = threadIdx.x;
  int jj = threadIdx.y;
    for (int w = 0; w < k; w += blockDim.x){
      sum = 0;
      A_s[ii*blockDim.y + jj] = A[i*k+jj+w];
      B_s[ii*blockDim.y + jj] = B[j+ii*n+w*n];
      __syncthreads();
      for (int h = 0; h < blockDim.x; h++) {
        sum += A_s[ii*blockDim.x + h] * B_s[h*blockDim.x + jj];
      }
      __syncthreads();
      C[i*n + j] += sum;
    }
}


extern "C" {
    void matmult_gpu6(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        // Initialize the output matrix with zeroes.
        hipMemset(d_C, 0, m*n * sizeof(double));
        dim3 BlockDim(16,16);
        dim3 NumBlocks((m-1)/16+1,(n-1)/16+1);
        m6<<<NumBlocks,BlockDim>>>(m, n, k, d_A, d_B, d_C);
        checkCudaErrors(hipDeviceSynchronize());

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
