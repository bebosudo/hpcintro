#include "hip/hip_runtime.h"
// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____           ____n____
//    |        |           |        |          |       |
//  m |    A   |   X    k  |    B   |  =    m  |   C   |
//    |        |           |        |          |       |
//    ---------            ---------           ---------
#include <hip/hip_runtime_api.h>
__global__ void m6(int m, int n, int k, double *A, double *B, double *C) {
  double sum;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;

  extern __shared__ double two_blocks[];
  __shared__ double* A_s;
  A_s = &two_blocks[0];
  __shared__ double* B_s;
  B_s = &two_blocks[blockDim.x*blockDim.y];

  int ii = threadIdx.x;
  int jj = threadIdx.y;
  const int blockdim = blockDim.x;

  for (int w = 0; w < k; w += blockDim.x){
      sum = 0;
      A_s[ii*blockdim + jj] = A[i*k+jj+w];
      B_s[ii*blockdim + jj] = B[j+ii*n+w*n];
    __syncthreads();
      for (int h = 0; h < blockdim; h++) {
        sum += A_s[ii*blockdim + h] * B_s[h*blockdim + jj];
      }
      __syncthreads();
      C[i*n + j] += sum;
  }
}


extern "C" {
    void matmult_gpu6(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        // Initialize the output matrix with zeroes.
        hipMemset(d_C, 0, m*n * sizeof(double));

        int bs = 16;
        dim3 blockDim(bs, bs);
        dim3 gridDim( (m-1)/blockDim.x+1, (n-1)/blockDim.y+1 );


        // https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
        // dynamically "pass" the shared memory to the kernel function.
        // Otherwise we should place some constants in the kernel function.
        m6<<<gridDim, blockDim, (blockDim.x*blockDim.y * 2 * sizeof(double))>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
