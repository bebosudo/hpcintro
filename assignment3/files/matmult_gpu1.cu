
#include <hip/hip_runtime.h>
// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____           ____n____
//    |        |           |        |          |       |
//  m |    A   |   X    k  |    B   |  =    m  |   C   |
//    |        |           |        |          |       |
//    ---------            ---------           ---------


__global__ void m1(int m, int n, int k, double *A, double *B, double *C) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            C[i*n + j] = 0;
        }
    }

    for (int i = 0; i < m; i++) {
        for (int h = 0; h < k; h++){
            for (int j = 0; j < n; j++) {
                C[i*n + j] += A[i*n + h] * B[h*n + j];
            }
        }
    }
}


extern "C" {
    void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);
        // cudaMemcpy(d_B, B, k*n * sizeof(double), cudaMemcpyHostToDevice);


        m1<<<1,1>>>(m, n, k, d_A, d_B, d_C);

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}