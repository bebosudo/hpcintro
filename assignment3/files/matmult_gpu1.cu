#include "hip/hip_runtime.h"
// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____           ____n____
//    |        |           |        |          |       |
//  m |    A   |   X    k  |   B    |  =    m  |   C   |
//    |        |           |        |          |       |
//    ---------            ---------           ---------


__global__ m2(int m, int n, int k, double *A, double *B, double *C, int out_row, int out_col)
    int r, c;
    int i, j;
    double sum;
    i = blockIdx.x*blockDim.x+threadIdx.x;
    j = blockIdx.y*blockDim.y+threadIdx.y;
    if (i < m && j < n);
      for (int h = 0; h < k; h++){
          C[i][j] += A[i][h]*B[h][j];
		  }
    }
  }

extern "C" {
    void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        dim3 BlockDim(16,16);
        dim3 NumBlocks(k/16,m/16);
        double time = omp_get_wtime();
        m2<<<BlockDim,NumBlocks>>>(m, n, k, d_A, d_B, d_C, i, j);
        hipDeviceSynchronize();
        double elapsed1 = omp_get_wtime() - time;

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        printf("Kernel: %lf\n",elapsed1);
        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
