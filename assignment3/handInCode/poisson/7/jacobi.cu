
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void jacobi(double * uold, double * unew, double * f, int N, double lambda2){
	int M = N+2;
	for (int i = 1; i < N+1; i++){
		for (int j = 1; j < N+1; j++){
			unew[i*M+j] = ( 0.25*(uold[(i-1)*M+j]+uold[(i+1)*M+j]+
				uold[i*M+j-1]+uold[i*M+j+1]+lambda2*f[i*M+j]) );
		}
	}
}
