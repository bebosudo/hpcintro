
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#define PRINT 1

__global__ void jacobi(double * uold, double * unew, double * f, int width, int height, double lambda2);

int main(int argc, char * argv[]){
	if (argc != 3){
		printf("Wrong number of argument (N kmax)");
		return -1;
	}

	int N = atoi(argv[1]);
	int M = N+2;
	int kmax = atoi(argv[2]);
	double lambda2 = (double)4/(M*M);

	// M*M room split into sections of M*Ha and M*Hb
	int HALF = M/2*M;	
	int HALF_1 = (M/2-1)*M;
	int H0 = (M/2) + 1;
	int H1 = 1+(M-1)/2 + 1;

	int size0 = M*H0*sizeof(double);
	int size1 = M*H1*sizeof(double);

	double *u1, *u2, *f;

	u1 =(double *)calloc(M*M,sizeof(double));
	u2 = (double *)calloc(M*M,sizeof(double));
	f = (double *)calloc(M*M,sizeof(double));
	if (u1 == NULL || u2 == NULL || f == NULL){
		printf("Memory allocation failed");
		return -1;
	}

	for(int i = 0; i < N+1; i++){
		u2[i*M] = 20;
		u2[N+1+M*i] = 20;
		u2[i] = 20;
		u1[i*M] = 20;
		u1[(N+1)+i*M] = 20;
		u1[i] = 20;
	}

	for (int i = 2*M/3; i <= 5*M/6; i++){
		for (int j = M/2; j <= 2*M/3; j++){
			f[i*M+j] = 200;
		}

	}

	double *d0_u1, *d0_u2, *d0_f;
	double *d1_u1, *d1_u2, *d1_f;

	hipSetDevice(0);
	hipDeviceEnablePeerAccess(1,0);
	hipMalloc( (void**)&d0_u1, size0);
	hipMalloc( (void**)&d0_u2, size0);
	hipMalloc( (void**)&d0_f, size0);

	hipMemcpy( d0_u1, u1, size0, hipMemcpyHostToDevice );
	hipMemcpy( d0_u2, u2, size0, hipMemcpyHostToDevice );
	hipMemcpy( d0_f, f, size0, hipMemcpyHostToDevice );

	hipSetDevice(1);
	hipDeviceEnablePeerAccess(0,0);
	hipMalloc( (void**)&d1_u1, size1);
	hipMalloc( (void**)&d1_u2, size1);
	hipMalloc( (void**)&d1_f, size1);

	hipMemcpy( d1_u1, &u1[HALF_1], size1, hipMemcpyHostToDevice );
	hipMemcpy( d1_u2, &u2[HALF_1], size1, hipMemcpyHostToDevice );
	hipMemcpy( d1_f, &f[HALF_1], size1, hipMemcpyHostToDevice );


	int blockSize = 32;
	dim3 dimBlock(blockSize,blockSize,1); 
  	int gridSize = 1 + ((M - 1) / (2*blockSize)); // M/(2*blockSize) round up.
  	dim3 dimGrid(gridSize,gridSize,1);

	double ts, te;
	ts = omp_get_wtime();
	for(int k = 0; k<kmax; k++){
		hipMemcpy(d1_u1, &d0_u1[HALF_1], M*sizeof(double),hipMemcpyDefault);
		hipMemcpy(&d0_u1[HALF], &d1_u1[M], M*sizeof(double),hipMemcpyDefault);

		// Update u2
		hipSetDevice(0);
		jacobi<<<dimGrid,dimBlock>>>(d0_u1,d0_u2,d0_f,N,H0-2,lambda2);
		hipSetDevice(1);
		jacobi<<<dimGrid,dimBlock>>>(d1_u1,d1_u2,d1_f,N,H1-2,lambda2);

		hipMemcpy(d1_u2, &d0_u2[HALF_1], M*sizeof(double),hipMemcpyDefault);
		hipMemcpy(&d0_u2[HALF], &d1_u2[M], M*sizeof(double),hipMemcpyDefault);

		// Update u1
		hipSetDevice(0);
		jacobi<<<dimGrid,dimBlock>>>(d0_u2,d0_u1,d0_f,N,H0-2,lambda2);
		hipSetDevice(1);
		jacobi<<<dimGrid,dimBlock>>>(d1_u2,d1_u1,d1_f,N,H1-2,lambda2);
	}
	hipDeviceSynchronize();

	te = omp_get_wtime() - ts;

	hipMemcpy(u1, d0_u1, HALF*sizeof(double), hipMemcpyDeviceToHost );
	hipMemcpy(&u1[HALF], &d1_u1[M], (M*M - HALF)*sizeof(double), hipMemcpyDeviceToHost );

	fprintf(stderr,"%s\n", hipGetErrorString(hipGetLastError()));
	printf("%d %d %lf\n",N,kmax,te);

	#if PRINT
	FILE *fp1 = fopen("results.txt","w");
	if (fp1 == NULL) {
		printf("Error opening file\n");
		return -1;
	}

	for (int i = 0; i < M; i++){
		for (int j = 0; j < M; j++){
			fprintf(fp1,"%.2lf ",u1[i*M+j]);
		}
		fprintf(fp1,"\n");
	}
	#endif

	hipFree(d0_u1);
	hipFree(d0_u2);
	hipFree(d0_f);
	hipFree(d1_u1);
	hipFree(d1_u2);
	hipFree(d1_f);
	free(u1);
	free(u2);
	free(f);

	return 0;
}
