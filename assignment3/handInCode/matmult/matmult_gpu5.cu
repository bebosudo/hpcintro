#include "hip/hip_runtime.h"
// #include <stdio.h>
#include <hip/hip_runtime_api.h>

// Write a first sequential implementation (matmult gpu1()) of matrix multiplication on the
// GPU that uses only a single thread. It should work for all matrix sizes. Hints:
// – You need CUDA code to allocate memory on the GPU, transfer A and B to the
//     GPU, transfer C back to the CPU, and free the allocated memory.
//
// Time your kernel for small matrix sizes and compare to the reference DGEMM on the CPU.

// matrix times matrix
// m represents the number of rows (the vertical length) of A and C,
// k represents the number of columns of A and the n. of rows of B,
// n represents the number of columns (the horizontal length) of B and C.
//    ____k____            ____n____          ____n____
//    |        |           |        |         |       |
//  m |   A    |    X    k |   B    |  =   m  |   C   |
//    |        |           |        |         |       |
//    ---------            ---------          ---------


__global__ void m5(int m, int n, int k, double *A, double *B, double *C) {
    // This variable 'two_blocks' (the name can be changed to whatever) comes
    // from the kernel invocation, and we have to "split" it manually into the
    // two variables we want to use.
    extern __shared__ double two_blocks[];
    __shared__ double* A_s;
    A_s = &two_blocks[0];
    __shared__ double* B_s;
    B_s = &two_blocks[blockDim.x*blockDim.y];

    int topleft_row_A = blockIdx.y*blockDim.y*k;
    int topleft_col_B = blockIdx.x*blockDim.x;

    // The blocks HAVE to have the same size, otherwise this matrix-matrix
    // mult on the small matrices cannot work.
    const int bl_side = blockDim.x;
    double sum;

    for (int w = 0; w < k; w += bl_side) {

        // We have to iterate over the two lines until reaching k.
        int topleft_row_A_curr_block = topleft_row_A + w;
        int topleft_col_B_curr_block = topleft_col_B + w*n;

        A_s[threadIdx.y*bl_side + threadIdx.x] = A[topleft_row_A_curr_block + threadIdx.y*k + threadIdx.x];
        // We just need each thread to load a single cell from the huge matrix
        // A & B, no matter if they don't load the same they are going to work on.
        B_s[threadIdx.y*bl_side + threadIdx.x] = B[topleft_col_B_curr_block + threadIdx.y*n + threadIdx.x];

        __syncthreads();

        sum = 0.0;
        for (int it=0; it < bl_side; it++) {
            sum += ( A_s[threadIdx.y*bl_side + it] * B_s[bl_side*it + threadIdx.x] );
        }

        // This second barrier syncronization is needed because there could be
        // some threads that could repeat the w_for loop and change A_s and B_s
        // while other are still reading from them.
        __syncthreads();

        // C[topleft_row_A_curr_block*n + topleft_col_B_curr_block + threadIdx.y*n + threadIdx.x] += sum;
        C[blockIdx.y*blockDim.y*n + threadIdx.y*n + blockIdx.x*blockDim.x + threadIdx.x] += sum;

    }
}


extern "C" {
    void matmult_gpu5(int m, int n, int k, double *A, double *B, double *C) {
        double* d_A, * d_B, * d_C;
        hipSetDevice(2);
        hipMalloc((void**)&d_A, m*k * sizeof(double));
        hipMalloc((void**)&d_B, k*n * sizeof(double));
        hipMalloc((void**)&d_C, m*n * sizeof(double));


        hipMemcpy(d_A, A, m*k * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, k*n * sizeof(double), hipMemcpyHostToDevice);

        // Initialize the output C matrix with zeroes.
        hipMemset(d_C, 0, m*n * sizeof(double));

        int bs = 4;
        dim3 blockDim(bs, bs);
        dim3 gridDim( (m-1)/blockDim.x+1, (n-1)/blockDim.y+1 );


        // https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
        // dynamically "pass" the shared memory to the kernel function.
        // Otherwise we should place some constants in the kernel function.
        m5<<<gridDim, blockDim, (blockDim.x*blockDim.y * 2 * sizeof(double))>>>(m, n, k, d_A, d_B, d_C);
        hipDeviceSynchronize();

        hipMemcpy(C, d_C, m*n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
