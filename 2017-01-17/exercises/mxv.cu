
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <omp.h>
// M is the length of the vector, while N is the other side of the (MxN)matrix.
#define M 5678
#define N 12340

// Matrix: MxN, Vector: Nx1, Result (vector): (MxN X Nx1) --> Mx1.

// matrix vector multiplication
void __global__ mxv(int m, int n, double *matrix, double *vector, double* vec_ret) {
    int i, j;
    double sum;

    for (i=0; i<m; i++) {
        sum = 0.0;
        for (j=0; j<n; j++) {
            sum += matrix[i*n+j] * vector[j];
        }
        vec_ret[i] = sum;
    }
}


int main(void){
    // The results are stored in A.
    double * mat = (double *)malloc(M*N * sizeof(double));
    double * vec = (double *)malloc(N * sizeof(double));
    double * vec_out = (double *)malloc(M * sizeof(double));

    double *d_mat, *d_vec, *d_vec_out;
    hipMalloc((void**)&d_mat, M*N * sizeof(double));
    hipMalloc((void**)&d_vec, N * sizeof(double));
    hipMalloc((void**)&d_vec_out, M * sizeof(double));

    // here we should check whether all these allocations are gone well.

    double time1 = omp_get_wtime();
    for (int i = 0; i < N*M; i++){
        mat[i] = 1;
    }
    for (int i = 0; i < N; i++){
        vec[i] = 1;
    }
    double elapsed1 = omp_get_wtime() - time1;

    // for (int i = 0; i < M; i++){
    //     vec_out[i] = 0;
    // }

    // we don't need to copy the output vector, since we are going to overwrite it.
    // cudaMemcpy(A_d, A, M*sizeof(double), cudaMemcpyHostToDevice);
    hipMemcpy(d_mat, mat, M*N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec, N * sizeof(double), hipMemcpyHostToDevice);

    // M is the length of the vector, while N is the other side of the matrix.
    // int m_size = M;
    // int n_size = N;
    // if (m_size > 1024) {
    //     m_size = 1024;
    // }

    double time2 = omp_get_wtime();
    mxv<<<4, 32>>>(M, N, d_mat, d_vec, d_vec_out);
    hipDeviceSynchronize();
    double elapsed2 = omp_get_wtime() - time2;

    double time3 = omp_get_wtime();
    hipMemcpy(vec_out, d_vec_out, M * sizeof(double), hipMemcpyDeviceToHost);
    double elapsed3 = omp_get_wtime() - time3;

    printf("memcpy HtoD: %lf\nmxv: %lf\nmemcpy DtoH: %lf\n\nTOTAL: %lf\n", elapsed1, elapsed2, elapsed3, elapsed1+elapsed2+elapsed3);

    // printf("vec_out[0] = %lf\n", vec_out[0]);
    // for (int i = 0; i<M; i++) {
    //     printf("%lf\n",vec_out[i]);
    // }

    free(mat); free(vec); free(vec_out);
    hipFree(d_mat); hipFree(d_vec); hipFree(d_vec_out);
}
